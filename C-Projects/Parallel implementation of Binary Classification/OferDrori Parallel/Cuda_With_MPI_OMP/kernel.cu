
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>



void freeCudaMemory(double* wCuda, int* resultCuda, double *pointsCuda);

hipError_t calculatePointsCuda(double* points, int N, int K, double * w, int *results);




__global__ void fCuda(double* points, int* N, int* K, double * w, int * results)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int i;
	int sign;
	if (id >= *N)
		return;
	double f = w[*K];
	for (i = 0; i < *K; i++)
	{
		f += points[id * (*K) + i] * w[i];
	}
	if (f >= 0)
		results[id] = 1;
	else
		results[id] = -1;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t calculatePointsCuda(double* points, int N, int K, double * w, int *results)
{
	char errorBuffer[100];
	double* pointsCuda;
	double * wCuda;
	int * resultCuda;
	int* K_cuda;
	int* N_cuda;
	hipError_t cudaStatus;
	hipDeviceProp_t props;


	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);

	hipGetDeviceProperties(&props, 0);

	cudaStatus = hipMalloc((void**)&K_cuda, sizeof(int));
	cudaStatus = hipMalloc((void**)&N_cuda, sizeof(int));

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&pointsCuda, sizeof(double)*N*K);
	//checkError(cudaStatus, cudaPoints, "cudaMalloc failed!");
	cudaStatus = hipMalloc((void**)&wCuda, sizeof(double)*(K+1));
	cudaStatus = hipMalloc((void**)&resultCuda, sizeof(int) * N);

	//	checkError(cudaStatus, cudaPoints, "cudaMalloc failed!");

	cudaStatus = hipMemcpy(K_cuda, &K, sizeof(int), hipMemcpyHostToDevice);

	cudaStatus = hipMemcpy(N_cuda, &N, sizeof(int), hipMemcpyHostToDevice);

	cudaStatus = hipMemcpy(pointsCuda, points, sizeof(double)*N*K, hipMemcpyHostToDevice);
	//checkError(cudaStatus, points, "cudaMemcpy failed!");
	cudaStatus = hipMemcpy(wCuda, w, sizeof(double)*(K+1), hipMemcpyHostToDevice);
	
	int numThreads = props.maxThreadsPerBlock < N ? props.maxThreadsPerBlock : N;
	int numBlocks = N / numThreads;
	int extraBlock = N % numThreads != 0;

	// Launch a kernel on the GPU with one thread for each element.
	fCuda <<<numBlocks + extraBlock, numThreads >>>(pointsCuda, N_cuda, K_cuda, wCuda, resultCuda);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	//sprintf(errorBuffer, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
	if (cudaStatus != hipSuccess)
	{
		printf(errorBuffer);
	}
	//checkError(cudaStatus, dev_c, errorBuffer);

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	//cudaStatus = cudaDeviceSynchronize();
	//sprintf(errorBuffer, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	//checkError(cudaStatus, dev_c, errorBuffer);

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(results, resultCuda, N * sizeof(int), hipMemcpyDeviceToHost);
	//checkError(cudaStatus, dev_c, "cudaMemcpy failed!");

	freeCudaMemory(wCuda, resultCuda, pointsCuda);

	return cudaStatus;
}

void freeCudaMemory(double* wCuda, int* resultCuda, double *pointsCuda)
{
	hipFree(wCuda);
	hipFree(resultCuda);
	hipFree(pointsCuda);
}



